/***************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <cutlass/wmma_matrix.h>
#ifdef CUTLASS_USE_WMMA_API

////////////////////////////////////////////////////////////////////////////////////////////////////

#include <cutlass/gemm/gemm.h>

#include <tools/test/perf/gemm/gemm_profiler.h>
#include <tools/test/perf/gemm/cutlass_dispatch.h>
#include <tools/test/perf/gemm/gemm_perf_testbed.h>
#include <cutlass/gemm/wmma_gemm_traits.h>

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Traits>
struct WmmaGemmDispatch {

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  WmmaGemmDispatch() {}

  /// Initializes params object
  WmmaGemmDispatch(int m, int n, int k, float alpha, half const* d_a, int lda,
                  half const* d_b, int ldb, float beta, float const* d_c, int ldc,
                  float* d_d, int ldd) {

    params.initialize(m, n, k, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  /// Initializes params object
  WmmaGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }

  /// Determines if problem is aligned (assuming no padding)
  static bool is_problem_aligned(
    int m,
    int n,
    int k) {

    bool aligned = true;

    if (kLayoutA == cutlass::MatrixLayout::kColumnMajor) {
      aligned = aligned && !(m % Gemm::Traits::GemmConfig::kScalarsPerLdgA);
    }
    else {
      aligned = aligned && !(k % Gemm::Traits::GemmConfig::kScalarsPerLdgA);
    }

    if (kLayoutB == cutlass::MatrixLayout::kColumnMajor) {
      aligned = aligned && !(k % Gemm::Traits::GemmConfig::kScalarsPerLdgB);
    }
    else {
      aligned = aligned && !(n % Gemm::Traits::GemmConfig::kScalarsPerLdgB);
    }

    aligned = aligned && !(m % Gemm::Traits::GemmConfig::kScalarsPerLdgC);

    return aligned;
  }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace perf {

////////////////////////////////////////////////////////////////////////////////////////////////////

int profile_wmma_gemm(TestbenchOutput &output, TestbenchOptions const &options) {

  typedef perf::GemmProfiler<cutlass::half_t, cutlass::half_t, float, float, float> GemmProfiler;

  int results = 0;

  if (!results) {

    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                        cutlass::MatrixLayout::kRowMajor>
    WmmaGemmTraits;

    typedef WmmaGemmDispatch<WmmaGemmTraits> Dispatch;

    profile_gemm<Dispatch, GemmProfiler>(output, "wmma_gemm_nt", options);
  }

  if (!results) {

    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                        cutlass::MatrixLayout::kColumnMajor>
    WmmaGemmTraits;

    typedef WmmaGemmDispatch<WmmaGemmTraits> Dispatch;

    profile_gemm<Dispatch, GemmProfiler>(output, "wmma_gemm_nn", options);
  }

  if (!results) {

    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                        cutlass::MatrixLayout::kColumnMajor>
      WmmaGemmTraits;

    typedef WmmaGemmDispatch<WmmaGemmTraits> Dispatch;

    profile_gemm<Dispatch, GemmProfiler>(output, "wmma_gemm_tn", options);
  }

  if (!results) {

    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                        cutlass::MatrixLayout::kRowMajor>
      WmmaGemmTraits;

    typedef WmmaGemmDispatch<WmmaGemmTraits> Dispatch;

    profile_gemm<Dispatch, GemmProfiler>(output, "wmma_gemm_tt", options);
  }

  return results;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace perf

////////////////////////////////////////////////////////////////////////////////////////////////////

#endif  // defined CUTLASS_USE_WMMA_API
