#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *   * Redistributions of source code must retain the above copyright notice, this list of
 *     conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above copyright notice, this list of
 *     conditions and the following disclaimer in the documentation and/or other materials
 *     provided with the distribution.
 *   * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *     to endorse or promote products derived from this software without specific prior written
 *     permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cutlass_unit_test.h>
#include <tools/util/host_tensor.h>
#include <tools/util/tensor_view_io.h>
#include <cutlass/shape.h>
#include <cutlass/predicate_vector.h>
#include <cutlass/tile_iterator.h>
#include <cutlass/tile_traits_standard.h>
#include <cutlass/iterator_access.h>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {

  template <typename Traits, typename Scalar>
  __global__ void load_store_global(
        typename cutlass::TileLoadIterator<Traits, Scalar, cutlass::IteratorAdvance::kH,
        cutlass::MemorySpace::kGlobal>::Scalar const *input,
        typename cutlass::TileStoreIterator<Traits, Scalar, cutlass::IteratorAdvance::kH,
        cutlass::MemorySpace::kGlobal>::Scalar *output
      ) {

    typedef cutlass::TileLoadIterator<Traits, Scalar, cutlass::IteratorAdvance::kH, cutlass::MemorySpace::kGlobal> LoadIterator;
    typedef cutlass::TileStoreIterator<Traits, Scalar, cutlass::IteratorAdvance::kH, cutlass::MemorySpace::kGlobal> StoreIterator;

    typename LoadIterator::Params load_params;
    typename StoreIterator::Params store_params;

    typedef typename Traits::Tile Tile;

    load_params.initialize(input, Tile::kH*Tile::kW, Tile::kW, 1);
    store_params.initialize(output, Tile::kH*Tile::kW, Tile::kW, 1);

    LoadIterator load_iterator(load_params);
    StoreIterator store_iterator(store_params);

    typename LoadIterator::Fragment fragment;

    load_iterator.load(fragment);
    store_iterator.store(fragment);
  }

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(TileIterator, tile_128x8_contiguous) {

  static int const M = 128;
  static int const N = 1;
  static int const K = 8;

  static int const kThreads = M;

  typedef cutlass::Shape<K, N, M> ThreadBlockTile;

  typedef cutlass::TileTraitsStandard<cutlass::Shape<N, K, M>, kThreads> Traits;

  cutlass::HostTensor<float> input;
  cutlass::HostTensor<float> output;

  input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
      cutlass::MatrixLayout::kColumnMajor);

  output.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
      cutlass::MatrixLayout::kColumnMajor);

  input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kW, 1));
  output.fill(0);

  test::load_store_global< Traits, float ><<<
    dim3(1,1,1),
    dim3(kThreads, 1)
  >>>(
    input.device_data(),
    output.device_data()
  );

  hipError_t result = hipDeviceSynchronize();
  ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                 << "\n";
  output.sync_host();

  EXPECT_TRUE(input.bit_equals(output));

}
////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(TileIterator, tile_128x8_rake) {

  static int const M = 128;
  static int const N = 1;
  static int const K = 8;

  static int const kThreads = 32;

  typedef cutlass::Shape<K, N, M> ThreadBlockTile;

  typedef cutlass::TileTraitsStandard<cutlass::Shape<N, K, M>, kThreads> Traits;

  cutlass::HostTensor<float> input;
  cutlass::HostTensor<float> output;

  input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
      cutlass::MatrixLayout::kColumnMajor);

  output.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
      cutlass::MatrixLayout::kColumnMajor);

  input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kW, 1));
  output.fill(0);

  test::load_store_global< Traits, float ><<<
    dim3(1,1,1),
    dim3(kThreads, 1)
  >>>(
    input.device_data(),
    output.device_data()
  );

  hipError_t result = hipDeviceSynchronize();
  ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                 << "\n";

  output.sync_host();

  EXPECT_TRUE(input.bit_equals(output));

}
////////////////////////////////////////////////////////////////////////////////////////////////////
}

