#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cutlass_unit_tests.h>
#include <tools/util/host_tensor.h>
#include <tools/util/tensor_view_io.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/shape.h>
#include <cutlass/gemm/sgemm_traits.h>
#include <cutlass/gemm/dgemm_traits.h>
#include <cutlass/gemm/hgemm_traits.h>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {

    // M/N/K struct.
    struct GemmDesc { 
      int m, n, k; 
      inline __host__ __device__ GemmDesc(int m_, int n_, int k_) : m(m_), n(n_), k(k_) {} 
    };

 /// Simple test to load from global memory and store to shared memory

    // Loading from global memory and storing to shared memory for A
    template <typename Traits>
    __global__ void gemm_load_global_store_shared_a(
        typename Traits::GlobalLoadStreamA::Scalar *output,
        typename Traits::GlobalLoadStreamA::Scalar const *input,
        int M,
        int N,
        int K,
        int ldm,
        int skew) {

        //Create shared memory.
        __shared__ typename Traits::SharedStorage shared_storage;

        // Create those iterators.
        typedef typename Traits::GlobalLoadStreamA GlobalLoadStreamA;

        typename GlobalLoadStreamA::Params global_load_params;
        GemmDesc desc(M, N, K);
        global_load_params.initialize(desc, input, ldm);

        GlobalLoadStreamA stream_a(global_load_params, shared_storage.main_loop.stream_a.global, M, N, K, cutlass::make_Coord(0, 0, 0));
        stream_a.copy();
        stream_a.commit();

        // store barrier
        __syncthreads();

        // one thread writes everything out
        if (threadIdx.x == 0) {
            for (int i = 0; i < (M+skew)*K; ++i) {
                output[i] = shared_storage.main_loop.stream_a.shared.scalars[i];
            }
        }

    }

    // Loading from global memory and storing to shared memory for B
    template <typename Traits>
    __global__ void gemm_load_global_store_shared_b(
        typename Traits::GlobalLoadStreamB::Scalar *output,
        typename Traits::GlobalLoadStreamB::Scalar const *input,
        int M,
        int N,
        int K,
        int ldm,
        int skew) {

        //Create shared memory.
        __shared__ typename Traits::SharedStorage shared_storage;

        // Create those iterators.
        typedef typename Traits::GlobalLoadStreamB GlobalLoadStreamB;
        typename GlobalLoadStreamB::Params global_load_params;
        GemmDesc desc(M, N, K);
        global_load_params.initialize(desc, input, ldm);

        GlobalLoadStreamB stream_b(global_load_params, shared_storage.main_loop.stream_b.global, M, N, K, cutlass::make_Coord(0, 0, 0));
        stream_b.copy();
        stream_b.commit();

        // store barrier
        __syncthreads();

        // one thread writes everything out
        if (threadIdx.x == 0) {
            for (int i = 0; i < (N+skew)*K; ++i) {
                output[i] = shared_storage.main_loop.stream_b.shared.scalars[i];
            }
        }

    }

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, A_float_contiguous) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::SgemmTraits<cutlass::MatrixLayout::kColumnMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            SgemmTraits;

    
    cutlass::HostTensor<float> input;
    cutlass::HostTensor<float> output;
    int skew = 0;

    input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);
    output.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);


    input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kW, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_a< SgemmTraits ><<<
        dim3(1,1,1),
        dim3(SgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        M,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";


    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, A_float_crosswise) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::SgemmTraits<cutlass::MatrixLayout::kRowMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            SgemmTraits;

    
    cutlass::HostTensor<float> input;
    cutlass::HostTensor<float> output;
    int skew = 4;

    input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kRowMajor);
    output.resize_matrix(ThreadBlockTile::kW + skew, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);


    input.fill_linear(cutlass::make_Coord(1, ThreadBlockTile::kD, 1, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_a< SgemmTraits ><<<
        dim3(1,1,1),
        dim3(SgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));

}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, B_float_contiguous) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::SgemmTraits<cutlass::MatrixLayout::kColumnMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            SgemmTraits;

    
    cutlass::HostTensor<float> input;
    cutlass::HostTensor<float> output;
    int skew = 0;

    input.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);
    output.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);


    input.fill_linear(cutlass::make_Coord(1, ThreadBlockTile::kH, 1, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_b< SgemmTraits ><<<
        dim3(1,1,1),
        dim3(SgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        N,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";


    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, B_float_crosswise) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::SgemmTraits<cutlass::MatrixLayout::kColumnMajor,
            cutlass::MatrixLayout::kColumnMajor, ThreadBlockTile >
            SgemmTraits;

    
    cutlass::HostTensor<float> input;
    cutlass::HostTensor<float> output;
    int skew = 4;

    input.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kColumnMajor);
    output.resize_matrix(ThreadBlockTile::kD + skew, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);


    input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kD, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_b< SgemmTraits ><<<
        dim3(1,1,1),
        dim3(SgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, A_double_contiguous) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::DgemmTraits<cutlass::MatrixLayout::kColumnMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            DgemmTraits;

    
    cutlass::HostTensor<double> input;
    cutlass::HostTensor<double> output;
    int skew = 0;

    input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);
    output.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);


    input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kW, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_a< DgemmTraits ><<<
        dim3(1,1,1),
        dim3(DgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        M,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";


    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, A_double_crosswise) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::DgemmTraits<cutlass::MatrixLayout::kRowMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            DgemmTraits;

    
    cutlass::HostTensor<double> input;
    cutlass::HostTensor<double> output;
    int skew = 2;

    input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kRowMajor);
    output.resize_matrix(ThreadBlockTile::kW + skew, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);


    input.fill_linear(cutlass::make_Coord(1, ThreadBlockTile::kD, 1, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_a< DgemmTraits ><<<
        dim3(1,1,1),
        dim3(DgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));

}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, B_double_contiguous) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::DgemmTraits<cutlass::MatrixLayout::kColumnMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            DgemmTraits;

    
    cutlass::HostTensor<double> input;
    cutlass::HostTensor<double> output;
    int skew = 0;

    input.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);
    output.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);


    input.fill_linear(cutlass::make_Coord(1, ThreadBlockTile::kH, 1, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_b< DgemmTraits ><<<
        dim3(1,1,1),
        dim3(DgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        N,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";


    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, B_double_crosswise) {

    static int const M = 64;
    static int const N = 64;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::DgemmTraits<cutlass::MatrixLayout::kColumnMajor,
            cutlass::MatrixLayout::kColumnMajor, ThreadBlockTile >
            DgemmTraits;

    
    cutlass::HostTensor<double> input;
    cutlass::HostTensor<double> output;
    int skew = 2;

    input.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kColumnMajor);
    output.resize_matrix(ThreadBlockTile::kD + skew, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);


    input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kD, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_b< DgemmTraits ><<<
        dim3(1,1,1),
        dim3(DgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, A_half_crosswise) {

    static int const M = 128;
    static int const N = 128;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::HgemmTraits<cutlass::MatrixLayout::kRowMajor, cutlass::MatrixLayout::kRowMajor, ThreadBlockTile >
            HgemmTraits;

    
    cutlass::HostTensor<cutlass::half_t> input;
    cutlass::HostTensor<cutlass::half_t> output;
    int skew = 8;

    input.resize_matrix(ThreadBlockTile::kW, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kRowMajor);
    output.resize_matrix(ThreadBlockTile::kW + skew, ThreadBlockTile::kD,
            cutlass::MatrixLayout::kColumnMajor);


    input.fill_linear(cutlass::make_Coord(1, ThreadBlockTile::kD, 1, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_a< HgemmTraits ><<<
        dim3(1,1,1),
        dim3(HgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();
    
    EXPECT_TRUE(input.bit_equals(output));

}

////////////////////////////////////////////////////////////////////////////////////////////////////
TEST(GemmSharedMemLayout, B_half_crosswise) {

    static int const M = 128;
    static int const N = 128;
    static int const K = 8;

    typedef cutlass::Shape<K, N, M> ThreadBlockTile;

    typedef cutlass::gemm::HgemmTraits<cutlass::MatrixLayout::kColumnMajor,
            cutlass::MatrixLayout::kColumnMajor, ThreadBlockTile >
            HgemmTraits;

    
    cutlass::HostTensor<cutlass::half_t> input;
    cutlass::HostTensor<cutlass::half_t> output;
    int skew = 8;

    input.resize_matrix(ThreadBlockTile::kD, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kColumnMajor);
    output.resize_matrix(ThreadBlockTile::kD + skew, ThreadBlockTile::kH,
            cutlass::MatrixLayout::kRowMajor);


    input.fill_linear(cutlass::make_Coord(1, 1, ThreadBlockTile::kD, 1));

    output.fill(0);

    test::gemm_load_global_store_shared_b< HgemmTraits ><<<
        dim3(1,1,1),
        dim3(HgemmTraits::kThreads, 1)
    >>>(
        output.device_data(),
        input.device_data(),
        M,
        N,
        K,
        K,
        skew
    );

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "\nCUDA kernel launch error: " << hipGetErrorString(result)
                                 << "\n";

    output.sync_host();

    EXPECT_TRUE(input.bit_equals(output));
}

////////////////////////////////////////////////////////////////////////////////////////////////////
}

